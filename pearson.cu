
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <string.h>

#define ISOLATE_LEN 188
#define NUM_REGIONS 2
#define LEN_23S 93
#define LEN_16S 95

extern "C" {

__global__ void pearson(uint32_t tileRowSize, float *isoDataRow,
                        uint32_t tileColSize, float *isoDataCol,
                        uint32_t tileRow, uint32_t tileCol,
                        uint32_t numIsolates, uint32_t simMatrixSize,
                        float *simMatrix) {
   //uint32_t iso_A_ndx = blockIdx.y * blockDim.y + threadIdx.y; // row
   //uint32_t iso_B_ndx = blockIdx.x * blockDim.x + threadIdx.x; // column

   // We don't want to compare isolates with themselves, or any comparisons
   // of a lower-numberes isolate to a higher-numbered one. Each pair of 
   // isolates (order doesn't matter) will only be compared once. This will
   // cause divergence only in the warps that lie along the main diagonal
   // of the comparison matrix.
   /*
   if ((tileRowSize * tileRow) + iso_A_ndx > 0 ||
       (tileColSize * tileCol) + iso_B_ndx > 0) { return;}
   if ((tileRowSize * tileRow) + iso_A_ndx >=
       (tileColSize * tileCol) + iso_B_ndx ||
       iso_A_ndx >= tileRowSize ||
       iso_B_ndx >= tileColSize) { return; }
       */

   /*
   // Initialize accumulators and the result.
   float pearson_sum = 0.0f;
   float peak_height_A = 0.0f, peak_height_B = 0.0f;
   float sum_A = 0.0f, sum_B = 0.0f, sum_AB = 0.0f,
         sum_A_squared = 0.0f, sum_B_squared = 0.0f;

   // Compute the sums for the 23-5 region (first 93).
   for (uint8_t ndx = 0; ndx < LEN_23S; ndx++) {
      peak_height_A = isoDataRow[iso_A_ndx * ISOLATE_LEN + ndx];
      peak_height_B = isoDataCol[iso_B_ndx * ISOLATE_LEN + ndx];

      sum_A += peak_height_A;
      sum_B += peak_height_B;
      sum_A_squared += peak_height_A * peak_height_A;
      sum_B_squared += peak_height_B * peak_height_B;
      sum_AB += peak_height_A * peak_height_B;
   }

   pearson_sum = (LEN_23S * sum_AB - sum_A * sum_B) /
                  sqrtf((LEN_23S * sum_A_squared - sum_A * sum_A) * 
                        (LEN_23S * sum_B_squared - sum_B * sum_B));

   peak_height_A = 0.0f, peak_height_B = 0.0f;
   sum_A = 0.0f, sum_B = 0.0f, sum_AB = 0.0f;
   sum_A_squared = 0.0f, sum_B_squared = 0.0f;

   // Compute the sums for the 16-23 region (last 95).
   for (uint8_t ndx = 0; ndx < LEN_16S; ndx++) {
      peak_height_A = isoDataRow[iso_A_ndx * ISOLATE_LEN + LEN_23S + ndx];
      peak_height_B = isoDataCol[iso_B_ndx * ISOLATE_LEN + LEN_23S + ndx];

      sum_A += peak_height_A;
      sum_B += peak_height_B;
      sum_A_squared += peak_height_A * peak_height_A;
      sum_B_squared += peak_height_B * peak_height_B;
      sum_AB += peak_height_A * peak_height_B;
   }

   pearson_sum += (LEN_16S * sum_AB - sum_A * sum_B) /
                   sqrtf((LEN_16S * sum_A_squared - sum_A * sum_A) * 
                         (LEN_16S * sum_B_squared - sum_B * sum_B));

   // Compute the index to store the result (a single dimensional array that is
   // a linear, packed representation of the upper half of the sim matrix)
   uint32_t result_ndx = (tileRowSize * tileRow) + iso_B_ndx - 1;
   for (uint32_t row_num = 1; row_num <= iso_A_ndx; row_num++) {
      result_ndx += (numIsolates - 1) - row_num;
   }

   if (result_ndx < simMatrixSize) {
      simMatrix[result_ndx] = pearson_sum / 2;
   }
   */

   printf("%d", 78);
   for (int i = 0; i < 10; i++) {
      simMatrix[i] = 99;
   }
   __syncthreads();
}

}

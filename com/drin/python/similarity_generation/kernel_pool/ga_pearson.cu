#include "hip/hip_runtime.h"
#include <stdint.h>
#include <string.h>

#define 16S_LEN 95
#define 23S_LEN 93

__global__ void pearson(uint32_t num_isolates, float **isolates,
                        uint32_t isolate_A, uint32_t isolate_B) {
   // Calculate relative <i, j> coords within this tile.
   uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; // column
   uint32_t j = blockIdx.y * blockDim.y + threadIdx.y; // row

   // Calculate the absolute <i, j> coords within the matrix.
   uint32_t i_abs = tile_col * tile_size + i;
   uint32_t j_abs = tile_row * tile_size + j;

   // We don't want to compare isolates with themselves, or any comparisons
   // of a lower-numberes isolate to a higher-numbered one. Each pair of 
   // isolates (order doesn't matter) will only be compared once. This will
   // cause divergence only in the warps that lie along the main diagonal
   // of the comparison matrix.
   if (i_abs <= j_abs)
      return;

   // Generate isolate |i_abs| and |j_abs|
   uint8_t i_allele_indices[kMaxAllelesPerIsolate]; 
   uint8_t j_allele_indices[kMaxAllelesPerIsolate]; 

   // Initialize accumulators and the result.
   float sum_x = 0, sum_y = 0, sum_x2 = 0, sum_y2 = 0, sum_xy = 0;

   // Compute the sums.
   for (int index = 0; index < length_alleles; ++index) {
      uint8_t x = 0, y = 0;

      x += alleles[i_allele_indices[alleleNdx] * length_alleles + index];
      y += alleles[j_allele_indices[alleleNdx] * length_alleles + index];

      sum_x += x;
      sum_y += y;
      sum_x2 += x * x;
      sum_y2 += y * y;
      sum_xy += x * y;
   }

   // Compute the Pearson coefficient using the "sometimes numerically
   // unstable" method because it's way more computationally efficient.
   float coeff = (length_alleles * sum_xy - sum_x * sum_y) /
      sqrtf((length_alleles * sum_x2 - sum_x * sum_x) * 
            (length_alleles * sum_y2 - sum_y * sum_y));

}

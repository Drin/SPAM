
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <string.h>

__device__ void dump_bucket(uint64_t *buckets,
      uint32_t num_ranges, uint32_t tile_size,
      uint32_t src_i, uint32_t src_j,
      uint32_t dest_i, uint32_t dest_j) {
   // Element-wise sum for each in 0 -> num_ranges.
   for (uint32_t k = 0; k < num_ranges; k++) {
      uint32_t src_index = (tile_size * tile_size * k) +
         (tile_size * src_i) + src_j;
      uint32_t dest_index = (tile_size * tile_size * k) +
         (tile_size * dest_i) + dest_j;
      buckets[dest_index] += buckets[src_index];
   }
}

__global__ void reduction(uint64_t *buckets, uint32_t num_ranges,
      uint32_t tile_size, uint32_t chunk_size) {
   // Calculate <i, j> coords within the tile.
   uint32_t i = blockIdx.x; // row
   uint32_t j = threadIdx.x * chunk_size; // column

   // Each chunk leader reduces its chunk.
   for (uint32_t k = 1; k < chunk_size; k++) {
      dump_bucket(buckets, num_ranges, tile_size, i, j + k, i, j);
   }

   // Wait for all the threads in this row to finish.
   __syncthreads();

   // Reduce each chunk leader into the zeroth element of the row.
   if (j == 0) {
      for (uint32_t k = 1; k < blockDim.x; k++) {
         dump_bucket(buckets, num_ranges, tile_size, i, k * chunk_size, i, 0);
      }
   }
}

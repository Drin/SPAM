
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <string.h>

#define ISOLATE_LEN 188
#define LEN_16S 95
#define LEN_23S 93

__global__ void pearson(uint32_t num_isolates, uint32_t tile_size,
                        uint32_t tile_row, uint32_t tile_col,
                        float *isolates, float *sim_matrix) {
   uint32_t iso_B_tile_ndx = blockIdx.x * blockDim.x + threadIdx.x; // column
   uint32_t iso_A_tile_ndx = blockIdx.y * blockDim.y + threadIdx.y; // row

   uint32_t iso_B_ndx = tile_row * tile_size + iso_B_tile_ndx;
   uint32_t iso_A_ndx = tile_col * tile_size + iso_A_tile_ndx;

   // We don't want to compare isolates with themselves, or any comparisons
   // of a lower-numberes isolate to a higher-numbered one. Each pair of 
   // isolates (order doesn't matter) will only be compared once. This will
   // cause divergence only in the warps that lie along the main diagonal
   // of the comparison matrix.
   if (iso_A_ndx >= iso_B_ndx ||
       iso_A_ndx >= num_isolates ||
       iso_B_ndx >= num_isolates) { return; }

   // Initialize accumulators and the result.
   float pearson_sum = 0.0f;
   float peak_height_A = 0.0f, peak_height_B = 0.0f;
   float sum_A = 0.0f, sum_B = 0.0f, sum_AB = 0.0f,
         sum_A_squared = 0.0f, sum_B_squared = 0.0f;

   // Compute the sums for the 23-5 region (first 93).
   for (uint8_t ndx = 0; ndx < LEN_23S; ndx++) {
      peak_height_A = isolates[iso_A_ndx * ISOLATE_LEN + ndx];
      peak_height_B = isolates[iso_B_ndx * ISOLATE_LEN + ndx];

      sum_A += peak_height_A;
      sum_B += peak_height_B;
      sum_A_squared += peak_height_A * peak_height_A;
      sum_B_squared += peak_height_B * peak_height_B;
      sum_AB += peak_height_A * peak_height_B;
   }

   pearson_sum = (LEN_23S * sum_AB - sum_A * sum_B) /
                  sqrtf((LEN_23S * sum_A_squared - sum_A * sum_A) * 
                        (LEN_23S * sum_B_squared - sum_B * sum_B));

   peak_height_A = 0.0f, peak_height_B = 0.0f;
   sum_A = 0.0f, sum_B = 0.0f, sum_AB = 0.0f;
   sum_A_squared = 0.0f, sum_B_squared = 0.0f;

   // Compute the sums for the 16-23 region (last 95).
   for (uint8_t ndx = 0; ndx < LEN_16S; ndx++) {
      peak_height_A = isolates[iso_A_ndx * ISOLATE_LEN + LEN_23S + ndx];
      peak_height_B = isolates[iso_B_ndx * ISOLATE_LEN + LEN_23S + ndx];

      sum_A += peak_height_A;
      sum_B += peak_height_B;
      sum_A_squared += peak_height_A * peak_height_A;
      sum_B_squared += peak_height_B * peak_height_B;
      sum_AB += peak_height_A * peak_height_B;
   }

   pearson_sum += (LEN_16S * sum_AB - sum_A * sum_B) /
                   sqrtf((LEN_16S * sum_A_squared - sum_A * sum_A) * 
                         (LEN_16S * sum_B_squared - sum_B * sum_B));

   // Compute the index to store the result (a single dimensional array that is
   // a linear, packed representation of the upper half of the sim matrix)
   uint32_t result_ndx = iso_B_ndx - 1;
   for (uint32_t row_num = 1; row_num <= iso_A_ndx; row_num++) {
      result_ndx += (num_isolates - 1) - row_num;
   }

   if (result_ndx < (num_isolates * num_isolates - 1) / 2) {
      sim_matrix[result_ndx] = pearson_sum / 2;
   }
}

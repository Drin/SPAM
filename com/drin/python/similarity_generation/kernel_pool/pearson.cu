#include "hip/hip_runtime.h"
#include <stdint.h>
#include <string.h>

#define ISOLATE_LEN 188
#define NUM_REGIONS 2
#define LEN_23S 93
#define LEN_16S 95

__constant__ uint8_t region_lens[LEN_23S, LEN_16S]
__constant__ uint8_t region_offsets[0, LEN_23S]

__global__ void cluster_pearson(uint32_t num_isolates_A, uint32_t *iso_list_A,
                                uint32_t num_isolates_B, uint32_t *iso_list_B,
                                uint32_t tsize, uint32_t trow, uint32_t tcol,
                                float *clust_sim) {
   uint32_t iso_B_tile_ndx = blockIdx.x * blockDim.x + threadIdx.x; // column
   uint32_t iso_A_tile_ndx = blockIdx.y * blockDim.y + threadIdx.y; // row

   uint32_t iso_B_ndx = trow * tsize + iso_B_tile_ndx;
   uint32_t iso_A_ndx = tcol * tsize + iso_A_tile_ndx;

   // Since we're comparing isolates in clusters we don't have to worry about
   // comparing isolates to themselves
   if (iso_A_ndx >= num_isolates_A || iso_B_ndx >= num_isolates_B) { return; }

   float pearson_sum = 0.0f;
   float peak_height_A = 0.0f, peak_height_B = 0.0f;
   float sum_A, sum_B, sum_AB, sum_A_squared, sum_B_squared;

   for (uint8_t reg_ndx = 0; reg_ndx < NUM_REGIONS; reg_ndx++) {
      // Initialize accumulators
      sum_A = 0.0f;
      sum_B = 0.0f;
      sum_AB = 0.0f;
      sum_A_squared = 0.0f;
      sum_B_squared = 0.0f;

      // Compute the sums for the current region (first 23-5 then 16-23)
      for (uint8_t ndx = 0; ndx < region_lens[reg_ndx]; ndx++) {
         peak_height_A = isolates[iso_A_ndx * ISOLATE_LEN + ndx +
                                  region_offsets[reg_ndx]];

         peak_height_B = isolates[iso_B_ndx * ISOLATE_LEN + ndx +
                                  region_offsets[reg_ndx]];

         sum_A += peak_height_A;
         sum_B += peak_height_B;
         sum_A_squared += peak_height_A * peak_height_A;
         sum_B_squared += peak_height_B * peak_height_B;
         sum_AB += peak_height_A * peak_height_B;
      }

      pearson_sum += (region_lens[reg_ndx] * sum_AB - sum_A * sum_B) /
               sqrtf((region_lens[reg_ndx] * sum_A_squared - sum_A * sum_A) * 
                     (region_lens[reg_ndx] * sum_B_squared - sum_B * sum_B));
   }

   atomicAdd(&clust_sim[0], pearson_sum / 2);
}

/*
__global__ void finder_pearson(uint32_t num_isolates_A, uint32_t *iso_list_A,
                               uint32_t num_isolates_B, uint32_t *iso_list_B,
                               uint32_t tsize, uint32_t trow, uint32_t tcol,
                               float *clust_sim) {
   uint32_t iso_B_tile_ndx = blockIdx.x * blockDim.x + threadIdx.x; // column
   uint32_t iso_A_tile_ndx = blockIdx.y * blockDim.y + threadIdx.y; // row

   uint32_t iso_B_ndx = trow * tsize + iso_B_tile_ndx;
   uint32_t iso_A_ndx = tcol * tsize + iso_A_tile_ndx;

   // Since we're comparing isolates in clusters we don't have to worry about
   // comparing isolates to themselves
   if (iso_A_ndx >= num_isolates_A || iso_B_ndx >= num_isolates_B) { return; }

   //find the appropriate isolate in the isolate data, which iso_list_A/B
   //contain an index to
   iso_B_ndx = iso_list_B[iso_B_ndx]; //go to the xth isolate to be compared
   iso_A_ndx = iso_list_A[iso_A_ndx]; //go to the yth isolate to be compared

   // Compute the index to store the result (a single dimensional array that is
   // a linear, packed representation of the upper half of the sim matrix)
   uint32_t result_ndx = iso_B_ndx - 1;
   for (uint32_t row_num = 1; row_num <= iso_A_ndx; row_num++) {
      result_ndx += (num_isolates - 1) - row_num;
   }

   if (result_ndx >= (num_isolates * num_isolates - 1) / 2) {
      return;
   }

   if (sim_matrix[result_ndx] == 0) {
      //TODO: Compute cluster similarities instead of isolate similarities
   }

   //This is commented out because the GTX 260 on my home machine cannot do
   //floating point atomic operations
   //atomicAdd(&clust_sim[0], sim_matrix[result_ndx]);
}
*/

__global__ void pearson(uint32_t num_isolates, uint32_t tile_size,
                        uint32_t tile_row, uint32_t tile_col,
                        float *isolates, float *sim_matrix) {
   uint32_t iso_B_tile_ndx = blockIdx.x * blockDim.x + threadIdx.x; // column
   uint32_t iso_A_tile_ndx = blockIdx.y * blockDim.y + threadIdx.y; // row

   uint32_t iso_B_ndx = tile_row * tile_size + iso_B_tile_ndx;
   uint32_t iso_A_ndx = tile_col * tile_size + iso_A_tile_ndx;

   // We don't want to compare isolates with themselves, or any comparisons
   // of a lower-numberes isolate to a higher-numbered one. Each pair of 
   // isolates (order doesn't matter) will only be compared once. This will
   // cause divergence only in the warps that lie along the main diagonal
   // of the comparison matrix.
   if (iso_A_ndx >= iso_B_ndx ||
       iso_A_ndx >= num_isolates ||
       iso_B_ndx >= num_isolates) { return; }

   // Initialize accumulators and the result.
   float pearson_sum = 0.0f;
   float peak_height_A = 0.0f, peak_height_B = 0.0f;
   float sum_A = 0.0f, sum_B = 0.0f, sum_AB = 0.0f,
         sum_A_squared = 0.0f, sum_B_squared = 0.0f;

   // Compute the sums for the 23-5 region (first 93).
   for (uint8_t ndx = 0; ndx < LEN_23S; ndx++) {
      peak_height_A = isolates[iso_A_ndx * ISOLATE_LEN + ndx];
      peak_height_B = isolates[iso_B_ndx * ISOLATE_LEN + ndx];

      sum_A += peak_height_A;
      sum_B += peak_height_B;
      sum_A_squared += peak_height_A * peak_height_A;
      sum_B_squared += peak_height_B * peak_height_B;
      sum_AB += peak_height_A * peak_height_B;
   }

   pearson_sum = (LEN_23S * sum_AB - sum_A * sum_B) /
                  sqrtf((LEN_23S * sum_A_squared - sum_A * sum_A) * 
                        (LEN_23S * sum_B_squared - sum_B * sum_B));

   peak_height_A = 0.0f, peak_height_B = 0.0f;
   sum_A = 0.0f, sum_B = 0.0f, sum_AB = 0.0f;
   sum_A_squared = 0.0f, sum_B_squared = 0.0f;

   // Compute the sums for the 16-23 region (last 95).
   for (uint8_t ndx = 0; ndx < LEN_16S; ndx++) {
      peak_height_A = isolates[iso_A_ndx * ISOLATE_LEN + LEN_23S + ndx];
      peak_height_B = isolates[iso_B_ndx * ISOLATE_LEN + LEN_23S + ndx];

      sum_A += peak_height_A;
      sum_B += peak_height_B;
      sum_A_squared += peak_height_A * peak_height_A;
      sum_B_squared += peak_height_B * peak_height_B;
      sum_AB += peak_height_A * peak_height_B;
   }

   pearson_sum += (LEN_16S * sum_AB - sum_A * sum_B) /
                   sqrtf((LEN_16S * sum_A_squared - sum_A * sum_A) * 
                         (LEN_16S * sum_B_squared - sum_B * sum_B));

   // Compute the index to store the result (a single dimensional array that is
   // a linear, packed representation of the upper half of the sim matrix)
   uint32_t result_ndx = iso_B_ndx - 1;
   for (uint32_t row_num = 1; row_num <= iso_A_ndx; row_num++) {
      result_ndx += (num_isolates - 1) - row_num;
   }

   if (result_ndx < (num_isolates * num_isolates - 1) / 2) {
      sim_matrix[result_ndx] = pearson_sum / 2;
   }
}
